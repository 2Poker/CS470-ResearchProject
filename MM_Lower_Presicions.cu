#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include "hip/hip_fp16.h"
#include "hip/hip_bf16.h"

#ifdef USE_FP16
     typedef __half Datatype;
#elif USE_BFLOAT16
    typedef __hip_bfloat16 Datatype;
#elif USE_FP32
    typedef float Datatype;
#elif USE_FP64
    typedef double Datatype;
#endif

__global__ void matrixMultiply(Datatype *A, Datatype *B, Datatype *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        // Datatype sum = __float2bDatatype(0.0f);
        Datatype sum = __float2half(0.0f);
        for (int k = 0; k < N; ++k) {
            Datatype a = A[row * N + k];
            Datatype b = B[k * N + col];
            sum += a * b;
        }
        C[row * N + col] = sum;
    }
}

int main(int argc, char *argv[]) {
    
    if (argc != 2) {
        fprintf(stderr, "Usage: [N]\n");
        return 1;
    }

    const long N = strtol(argv[1], NULL, 10); 
    size_t bytes = N * N * sizeof(Datatype);
    printf("size=%lu\n", sizeof(Datatype));

    // Host matrices
    Datatype *h_A = new Datatype[N * N];
    Datatype *h_B = new Datatype[N * N];
    Datatype *h_C = new Datatype[N * N];

    // Initialize input matrices A and B
    for (int i = 0; i < N * N; ++i) {

        #ifdef USE_FP16
            h_A[i] = __float2half((float)rand() / RAND_MAX); 
            h_B[i] = __float2half((float)rand() / RAND_MAX); 
        #elif USE_BFLOAT16
            h_A[i] = __float2bfloat16((float)rand() / RAND_MAX); 
            h_B[i] = __float2bfloat16((float)rand() / RAND_MAX); 
        #elif USE_FP32
            h_A[i] = (float)rand() / RAND_MAX; 
            h_B[i] = (float)rand() / RAND_MAX;
        #elif USE_FP64
            h_A[i] = (double)rand() / RAND_MAX; 
            h_B[i] = (double)rand() / RAND_MAX;
        #endif
    }

    // Device matrices
    Datatype *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    // Perform warm-up to ensure data is in GPU memory
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Start the timer
    // auto start = std::chrono::steady_clock::now();
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

    // Launch the kernel for actual computation
    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    hipEventRecord(stop);
	hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("[+] GPU(without Tensor Cores) Elapsed Time: %f ms\n", milliseconds);

    // Synchronize to ensure all kernels are finished
    

    // Stop the timer
    // auto end = std::chrono::steady_clock::now();
    // std::chrono::duration<double> elapsed_seconds = end - start;
    // std::cout << "Elapsed time: " << elapsed_seconds.count() * 1000 << " milliseconds" << std::endl;

    printf("Matrix multiplication completed\n");
    // Allocate host memory for the result matrix
    Datatype *h_C_result = new Datatype[N * N];

    // Copy result matrix from device to host
    hipMemcpy(h_C_result, d_C, N * N * sizeof(Datatype), hipMemcpyDeviceToHost);

    // Print the result matrix
    // for(int i = 0; i < N; ++i)
    // {
    //     for(int j = 0; j < N; ++j)
    //     {
    //         printf("%.20f ", __float2half(h_C_result[i * N + j]));
            
    //     }
    //     printf("\n");
    // }

    // Free the host memory for the result matrix
    delete[] h_C_result;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
